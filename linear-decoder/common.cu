#include "hip/hip_runtime.h"
#include "common.cuh"
/**
 *   Cublas GPU matrix multiplication
 */

int gpu_blas_dot(const Matrix &src1, const Matrix &src2,hipblasHandle_t *handle, cudaPrecision *result)
{
    hipblasStatus_t hipblasStatus_t;

    if (src1.row != src2.row || src1.col != src2.col)
    {
        fprintf(stderr, "gpu_blas_dot error: dimension not compatible\n");
        return -1;
    }

    // hipblasStatus_t = hipblasDdot(*handle, src1.row * src1.col, src1.elements, 1, src2.elements, 1, result);
    hipblasStatus_t = hipblasSdot(*handle, src1.row * src1.col, src1.elements, 1, src2.elements, 1, result);

    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "hipblasSaxpy error\n");
        return -1;
    }
    return 0;
}

int gpu_blas_nrm2(const Matrix &src, hipblasHandle_t *handle, cudaPrecision *result)
{
    hipblasStatus_t hipblasStatus_t;

    // hipblasStatus_t = hipblasDnrm2(*handle, src.row * src.col, src.elements, 1, result);
    hipblasStatus_t = hipblasSnrm2(*handle, src.row * src.col, src.elements, 1, result);

    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "hipblasSaxpy error\n");
        return -1;
    }
    return 0;
}

// dst = alpha * src + dst
int gpu_blas_axpy(Matrix &dst, const Matrix &src, hipblasHandle_t *handle, cudaPrecision alpha /*= 1*/)
{
    hipblasStatus_t hipblasStatus_t;

    if (dst.row != src.row || dst.col != src.col)
    {
        fprintf(stderr, "gpu_blas_minus error: dimension not compatible\n");
        return -1;
    }

    // hipblasStatus_t = hipblasDaxpy(*handle, dst.row * dst.col, &alpha, src.elements, 1, dst.elements, 1);
    hipblasStatus_t = hipblasSaxpy(*handle, dst.row * dst.col, &alpha, src.elements, 1, dst.elements, 1);

    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "hipblasSaxpy error\n");
        return -1;
    }
    return 0;
}

int gpu_blas_mcopy(Matrix &dst, const Matrix &src, hipblasHandle_t *handle)
{
    hipblasStatus_t hipblasStatus_t;

    if (dst.row != src.row || dst.col != src.col)
    {
        fprintf(stderr, "gpu_blas_mcopy error: dimension not compatible\n");
        return -1;
    }

    // hipblasStatus_t = hipblasDcopy(*handle, dst.row * dst.col, src.elements, 1, dst.elements, 1);
    hipblasStatus_t = hipblasScopy(*handle, dst.row * dst.col, src.elements, 1, dst.elements, 1);

    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "hipblasScopy error\n");
        return -1;
    }
    return 0;
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
int gpu_blas_mmul(const Matrix &C, const Matrix &A, const Matrix &B,
        hipblasOperation_t transa, hipblasOperation_t transb,
        hipblasHandle_t *handle, cudaPrecision alf /* = 1*/, cudaPrecision bet /* = 0*/)
{
    int lda = A.row;
    int ldb = B.row;
    int ldc = C.row;

    int m, n, ka, kb;

    const cudaPrecision *alpha = &alf;
    const cudaPrecision *beta = &bet;
    hipblasStatus_t hipblasStatus_t;
    hipError_t cudaStatus;

    if (transa == HIPBLAS_OP_N)
    {
        m = A.row;
        ka = A.col;
    }
    else
    {
        m = A.col;
        ka = A.row;
    }

    if (transb == HIPBLAS_OP_N)
    {
        n = B.col;
        kb = B.row;
    }
    else
    {
        n = B.row;
        kb = B.col;
    }

    if (ka != kb)
    {
        fprintf(stderr, "gpu_blas_mmul error: dimension not compatible: (%d, %d)\n", ka, kb);
        return -1;
    }

    //fprintf(stderr, "(%d, %d) = (%d, %d) * (%d %d)\n", C.row, C.col, A.row, A.col, B.row, B.col);

    // hipblasStatus_t = hipblasDgemm(*handle, transa, transb, m, n, ka, alpha, A.elements, lda, B.elements, ldb, beta, C.elements, ldc);
    hipblasStatus_t = hipblasSgemm(*handle, transa, transb, m, n, ka, alpha, A.elements, lda, B.elements, ldb, beta, C.elements, ldc);

    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "hipblasSgemm error\n");
        return -1;
    }

    /*
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        return -1;
    }*/

    return 0;
}

// Z(nHidden, nSample) = logistic(Z(nHidden, nSample) + B(nHidden))
// in place update
__global__ void biasAndLogistic(Matrix Z, Matrix b)
{
  cudaPrecision z;

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < Z.row && col < Z.col)
  {
      z = Z.elements[IDX2C(row, col, Z.row)]; // column major
      z += b.elements[row];
      //z = 1.0f / (1.0f + exp(-z)); // fast-math
      z = 1.0f / (1.0f + __expf(-z)); // fast-math

      Z.elements[IDX2C(row, col, Z.row)] = z;
  }
}

// Z(nHidden, nSample) = logistic(Z(nHidden, nSample) + B(nHidden))
// in place update
__global__ void bias(Matrix Z, Matrix b)
{
    cudaPrecision z;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < Z.row && col < Z.col)
    {
        z = Z.elements[IDX2C(row, col, Z.row)];
        z += b.elements[row];

        Z.elements[IDX2C(row, col, Z.row)] = z;
    }
}

// ann bp scaling delta2 = (delta2 + sparsity_der) .* a2 .* (1 - a2)
__global__ void annScaling(Matrix delta2, Matrix sparsity_der, Matrix a2)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < delta2.row && col < delta2.col)
    {
        int offset = IDX2C(row, col, delta2.row);

        cudaPrecision delta2_ = delta2.elements[offset];
        cudaPrecision sparsity_der_ = sparsity_der.elements[row];
        cudaPrecision a2_ = a2.elements[offset];

        delta2.elements[offset] = (delta2_ + sparsity_der_) * a2_ * (1 - a2_);
    }
}

// sparsity_der = beta * (- sparsity ./ rho + (1 - sparsity) ./ (1 - rho));
// sparsity derivative
__global__ void dSparsity(Matrix sparsity_der, Matrix rho, cudaPrecision beta, cudaPrecision sparsity)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rho.row && col < rho.col)
    {
        int offset = IDX2C(row, col, rho.row);

        cudaPrecision rho_ = rho.elements[offset];
        sparsity_der.elements[offset] = beta * (- sparsity / rho_ + (1 - sparsity) / (1 - rho_));
        // sparsity_der.elements[offset] = 0;
    }
}

// KL div, store elements into matrix
// div = sparsty .* log(sparsty ./ rho) + (1 - sparsty) .* log((1 - sparsty) ./ (1 - rho));
// save into rho
__global__ void costSparsity(Matrix rho, cudaPrecision sparsity)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rho.row && col < rho.col)
    {
        int offset = IDX2C(row, col, rho.row);

        cudaPrecision rho_ = rho.elements[offset];
        // cudaPrecision div = sparsity * log(sparsity / rho_) + (1 - sparsity) * log((1 - sparsity) / (1 - rho_)); // __logf: fast-math
        cudaPrecision div = sparsity * __logf(sparsity / rho_) + (1 - sparsity) * __logf((1 - sparsity) / (1 - rho_)); // __logf: fast-math

        rho.elements[offset] = div;
    }
}
